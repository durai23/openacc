#include <hip/hip_runtime.h>

__global__ void cudaVecAdd(
	double * __restrict__ a, 
	double * __restrict__ b, 
	double * __restrict__ c, 
	int n
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n) {
        // TODO: Calculate c = a + b for element i
        //
        // Hint: This is as easy as it sounds
    }
}

extern "C" void cudaVecAddWrapper(
	double * __restrict__ a, 
	double * __restrict__ b, 
	double * __restrict__ c, 
	int n,
	int nThreads,
	int nBlocks
) {
	// TODO: Call cudaVecAdd with nBlocks blocks and nThreads threads
}
