#include <hip/hip_runtime.h>

#include <thrust/device_ptr.h>
#include <thrust/reduce.h>

extern "C" void thrustReduceWrapper(
	double * __restrict__ c,
	int n,
	float * sum
) {
	thrust::device_ptr<double> c_ptr = thrust::device_pointer_cast(c);
	// TODO: Call thrust::reduce using c_ptr (the length is n) and give thrust's output to the calling function by means of sum
	// Hint: thrust::reduce can use device_ptr address as InputIterators
	// Documentation: https://thrust.github.io/doc/group__reductions.html#ga69434d74f2e6117040fb38d1a28016c2
}
