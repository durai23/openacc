#include <hip/hip_runtime.h>

#include <thrust/device_ptr.h>
#include <thrust/reduce.h>

extern "C" void thrustReduceWrapper(
	double * __restrict__ c,
	int n,
	float * sum
) {
	thrust::device_ptr<double> c_ptr = thrust::device_pointer_cast(c);
	*sum = thrust::reduce(c_ptr, c_ptr + n);
}
