#include <hip/hip_runtime.h>

__global__ void cudaVecAdd(
	double * __restrict__ a, 
	double * __restrict__ b, 
	double * __restrict__ c, 
	int n
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

extern "C" void cudaVecAddWrapper(
	double * __restrict__ a, 
	double * __restrict__ b, 
	double * __restrict__ c, 
	int n,
	int nThreads,
	int nBlocks
) {
	cudaVecAdd<<<nBlocks, nThreads>>>(a, b, c, n);
}
